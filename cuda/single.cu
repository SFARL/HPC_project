#include <stdio.h>
#include <math.h>
#include <iostream>
#include <stdlib.h>
#include <fstream>
#include <chrono>
#include <iomanip>
#include <algorithm>
#include <random>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <hiprand/hiprand.h>
#include <string>
#include <string.h>
using namespace std;

const int bfs_size = 128;
// nvcc single.cu -o single.o
// Function to prlong the Sudoku grid
void read_grid(int *grid, int n, string filename)
{
    ifstream file(filename);
    if (file.is_open())
    {
        for (int i = 0; i < n * n; i++)
        {
            file >> grid[i];
        }
        file.close();
    }
    else
    {
        cout << "Error: Unable to open input file." << endl;
    }
}

// Function to prlong the Sudoku grid
void print_grid(int *grid, int n)
{
    for (int i = 0; i < n * n; i++)
    {
        cout << std::setw(2) << grid[i] << " ";
        if ((i + 1) % n == 0)
            cout << endl;
    }
}
__global__ void
cudaBFS(int *old_grid, int *new_grid, int grid_total, int *grid_index, int *empty_space, int *empty_cnt, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    while (index < grid_total)
    {
        bool found = false;
        int start = index * n * n;
        // from the start of the grid to find next avaiable space
        for (int i = start; i < (start + n * n) && !found; i++)
        {
            if (old_grid[i] == 1)
            {
                continue;
            }
            found = true;
            int row = (i - start) / n;
            int col = (i - start) % n;

            // check avaible number
            for (int ava = 1; ava <= n; ava++)
            {

                // check row and col
                for (int j = 0; j < n; j++)
                {
                    if (old_grid[start + row * n + j] == ava)
                    {
                        break;
                    }
                    if (old_grid[start + j * n + col] == ava)
                    {
                        break;
                    }
                }

                // check board
                for (int j = 0; j < sqrtf(n); j++)
                {
                    for (int k = 0; k < sqrtf(n); k++)
                    {
                        if (old_grid[start + (row / n + j) * n + (col / n + k)] == ava)
                        {
                            break;
                        }
                    }
                }

                // if avaible, add to new grid
                int next_grid_index = atomicAdd(grid_index, 1);
                if (next_grid_index >= bfs_size)
                {
                    return;
                }
                int empty_index = 0;
                // This step maybe refact to using old gird index information
                for (int r = 0; r < n; row++)
                {
                    for (int c = 0; c < n; col++)
                    {
                        new_grid[next_grid_index * n * n + r * n + c] = old_grid[index * n * n + r * n + c];

                        // calculate the new grid empty space.
                        if (old_grid[index * n * n + r * n + c] == 0 && (r != row || c != col))
                        {
                            empty_space[next_grid_index * n * n + empty_index] = r * n + c;
                            empty_index++;
                        }
                    }
                }
                empty_cnt[next_grid_index] = empty_index;
                new_grid[next_grid_index * n * n + row * n + col] = ava;
            }
        }
        index += blockDim.x * gridDim.x;
    }
}

__device__ bool is_safe(int *grid, int n, int index)
{
    int row = index / n;
    int col = index % n;
    int num = grid[index];
    // check row and col
    for (int i = 0; i < n; i++)
    {
        if (grid[row * n + i] == num && i != col)
        {
            return false;
        }
        if (grid[i * n + col] == num && i != row)
        {
            return false;
        }
    }

    // check board
    for (int i = 0; i < sqrtf(n); i++)
    {
        for (int j = 0; j < sqrtf(n); j++)
        {
            if (grid[(row / n + i) * n + (col / n + j)] == num && (row / n + i) * n + (col / n + j) != index)
            {
                return false;
            }
        }
    }
    return true;
}

__global__ void cudaBackTrack(int *grid, int grid_total, int *emptySpaces, int *empty_cnt, bool *finished, int *solved, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int *grid_now, *empty_now, empty_cnt_now;
    while (!(*finished) && index < grid_total)
    {
        int empty_index = 0;
        grid_now = grid + index * n * n;
        empty_now = emptySpaces + index * n * n;
        empty_cnt_now = empty_cnt[index];
        while (empty_index < empty_cnt_now)
        {
            grid_now[empty_now[empty_index]]++;
            if (is_safe(grid_now, n, empty_now[empty_index]))
            {
                empty_index++;
            }
            else
            {
                if (grid_now[empty_now[empty_index]] == n)
                {
                    grid_now[empty_now[empty_index]] = 0;
                    empty_index--;
                }
            }
        }
        if (empty_index == empty_cnt_now)
        {
            *finished = true;
            for (int i = 0; i < n * n; i++)
            {
                solved[i] = grid_now[i];
            }
        }
        index += blockDim.x * gridDim.x;
    }
}
void callCudaBFS(int gridDim, int blockDim, int *old_grid, int *new_grid, int grid_total, int *grid_index, int *empty_space, int *empty_cnt, int n)
{
    cudaBFS<<<gridDim, blockDim>>>(old_grid, new_grid, grid_total, grid_index, empty_space, empty_cnt, n);
}

void callCudaBackTrack(int gridDim, int blockDim, int *grid, int grid_total, int *emptySpaces, int *empty_cnt, bool *finished, int *solved, int n)
{
    cudaBackTrack<<<gridDim, blockDim>>>(grid, grid_total, emptySpaces, empty_cnt, finished, solved, n);
}
int main()
{
    // if (argc < 6)
    // {
    //     printf("Usage: ./sequential <int:n> <string:input_file> <string:answer_file> <int:gridDim> <int:blockDim> \n");
    //     abort();
    // }
    // int root = atoi(argv[1]);
    // string input_filename = argv[2];
    // string answer_filename = argv[3];
    // int gridDim = atoi(argv[4]);
    // int blockDim = atoi(argv[5]);
    int root = 3;
    string input_filename = "../9.txt";
    string answer_filename = "../9_answer.txt";
    int gridDim = 4;
    int blockDim = 4;
    int n = root * root;
    int *grid = (int *)malloc(n * n * sizeof(int));
    read_grid(grid, n, input_filename);
    cout << "Input grid: " << endl;
    print_grid(grid, n);
    // Prepare for the gpu parallel
    int *new_grid, *old_grid;
    int *empty_space, *empty_cnt;
    int *grid_index;
    int grid_total = 1;

    int bfs_array_size = bfs_size * n * n * sizeof(int); // initial bfs search 128 grid

    // cuda allocate memory
    hipMalloc(&new_grid, bfs_array_size);
    hipMalloc(&old_grid, bfs_array_size);
    hipMalloc(&empty_space, bfs_array_size);
    hipMalloc(&empty_cnt, bfs_array_size / n / n + 1);
    hipMalloc(&grid_index, sizeof(int));

    // initialize the grid to 0
    hipMemset(grid_index, 0, sizeof(int));
    hipMemset(new_grid, 0, bfs_array_size);
    hipMemset(old_grid, 0, bfs_array_size);

    hipMemcpy(old_grid, grid, n * n * sizeof(int), hipMemcpyHostToDevice);

    cudaBFS<<<gridDim, blockDim>>>(old_grid, new_grid, grid_total, grid_index, empty_space, empty_cnt, n);

    int *tempPointer = old_grid;
    while (grid_total < bfs_size)
    {
        hipMemcpy(&grid_total, grid_index, sizeof(int), hipMemcpyDeviceToHost);
        hipMemset(grid_index, 0, sizeof(int));

        callCudaBFS(gridDim, blockDim, old_grid, new_grid, grid_total, grid_index, empty_space, empty_cnt, n);

        tempPointer = old_grid;
        old_grid = new_grid;
        new_grid = tempPointer;
    }
    hipMemcpy(&grid_total, grid_index, sizeof(int), hipMemcpyDeviceToHost);

    bool *finished;
    int *solved;
    hipMalloc(&solved, n * n * sizeof(int));
    hipMalloc(&finished, sizeof(bool));

    hipMemcpy(solved, old_grid, n * n * sizeof(int), hipMemcpyDeviceToDevice);
    hipMemset(finished, false, sizeof(bool));

    callCudaBackTrack(gridDim, blockDim, old_grid, grid_total, empty_space, empty_cnt, finished, solved, n);
    hipStreamSynchronize(0);
    print_grid(solved, n);

    delete[] grid;
    delete[] solved;
    hipFree(new_grid);
    hipFree(old_grid);
    hipFree(empty_space);
    hipFree(empty_cnt);
    hipFree(grid_index);
    hipFree(finished);
    hipFree(solved);
    return 0;
}